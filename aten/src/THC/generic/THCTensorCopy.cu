
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/THCTensorCopy.cu"
#else

void THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  if (dst == src) return;
  THC_copyTensor<scalar_t, scalar_t>(state, dst, src);
}

template <>
THCTensor *THCTensor_newClone<scalar_t>(THCState *state, THCTensor *self) {
  THCTensor* tensor =
      THCTensor_new(state, THTensor_getStoragePtr(self)->dtype());
  THCTensor_resizeAs(state, tensor, self);
  THC_copyTensor<scalar_t, scalar_t>(state, tensor, self);
  return tensor;
}

template <>
THCTensor *THCTensor_newContiguous<scalar_t>(THCState *state, THCTensor *self)
{
  if(!self->is_contiguous()) {
    return THCTensor_newClone<scalar_t>(state, self);
  } else {
    THCTensor_retain(state, self);
    return self;
  }
}


template <>
void THCTensor_freeCopyTo<scalar_t>(THCState *state, THCTensor *self, THCTensor *dst) {
  if(self != dst)
    THC_copyTensor<scalar_t, scalar_t>(state, dst, self);

  THCTensor_free(state, self);
}

template <>
void THCTensor_copyIgnoringOverlaps<scalar_t>(THCState* state, THCTensor* dst, THCTensor* src) {
  // Called when we are copying into an overlapping index `dst`, but
  // we don't care which writer wins. Hacky but it works.
  // This is itself invoked by pointwiseApply2 / THCTensor_copy in
  // case that there are write overlaps.
  // FIXME: really, overlapping writes should be illegal/an error in Torch
  THC_pointwiseApply2<scalar_t, scalar_t>(
    state, dst, src,
    CopyOp<scalar_t, scalar_t>(),
    ReadOnly, /* ignore overwrites */
    ReadOnly);
}

void THCTensor_(copyIgnoringOverlaps)(THCState* state, THCTensor* dst, THCTensor* src) {
  THCTensor_copyIgnoringOverlaps<scalar_t>(state, dst, src);
}

#define IMPLEMENT_THC_CUDA_TENSOR_COPY(TYPEC, TYPECUDA, SCALARC)        \
  void THCTensor_(copyCuda##TYPEC)(THCState *state,                     \
                                   THCTensor *self,                     \
                                   THCuda##TYPECUDA##Tensor *src) {     \
    THC_copyTensor<scalar_t, SCALARC>(state, self, src); \
  }

IMPLEMENT_THC_CUDA_TENSOR_COPY(Byte, Byte, uint8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Char, Char, int8_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Short, Short, int16_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Int, Int, int32_t)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Long, Long, int64_t)
// THCudaTensor aka the non-existent THCudaFloatTensor
IMPLEMENT_THC_CUDA_TENSOR_COPY(Float, , float)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Double, Double, double)
IMPLEMENT_THC_CUDA_TENSOR_COPY(Half, Half, at::Half)

#undef IMPLEMENT_THC_CUDA_TENSOR_COPY

#endif
