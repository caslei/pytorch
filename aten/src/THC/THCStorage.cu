#include "THCStorage.hpp"

#include "THCThrustAllocator.cuh"
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#if CUDA_VERSION >= 7000 || defined(__HIP_PLATFORM_HCC__)
#include <thrust/system/cuda/execution_policy.h>
#endif

#include "TH/THHalf.h"

#include "generic/THCStorage.cu"
#include "THCGenerateAllTypes.h"
