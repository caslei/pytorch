#include "THCUNN.h"
#include "TH/THHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCTensor.hpp"

#include <hipsparse.h>

static hipsparseHandle_t cusparse_handle = 0;

static void init_cusparse() {
  if (cusparse_handle == 0) {
    hipsparseStatus_t status = hipsparseCreate(&cusparse_handle);
    if (status != HIPSPARSE_STATUS_SUCCESS) {
      THError("CUSPARSE Library initialization failed");
    }
  }
}

void THNN_CudaHalfSparseLinear_updateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_accGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          float weightDecay,
          float scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyUpdateOutput(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *output,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_legacyAccGradParameters(
          THCState *state,
          THCudaHalfTensor *input,
          THCudaHalfTensor *gradOutput,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          float weightDecay,
          float scale) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_zeroGradParameters(
          THCState *state,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

void THNN_CudaHalfSparseLinear_updateParameters(
          THCState *state,
          THCudaHalfTensor *weight,
          THCudaHalfTensor *bias,
          THCudaHalfTensor *gradWeight,
          THCudaHalfTensor *gradBias,
          THCudaHalfTensor *lastInput,
          float learningRate) {
  THError("THCudaHalfTensor not supported with SparseLinear");
}

#include "generic/SparseLinear.cu"
#include "THCGenerateFloatType.h"
#include "generic/SparseLinear.cu"
#include "THCGenerateDoubleType.h"
